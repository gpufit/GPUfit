#include "hip/hip_runtime.h"
#include "gpu_data.cuh"
#include <hip/hip_runtime.h>
#include <>

GPUData::GPUData(Info const & info) :
    chunk_size_(0),
    info_(info),

    data_( info_.max_chunk_size_*info_.n_points_ ),
    weights_( info_.use_weights_ ? info_.n_points_ * info_.max_chunk_size_ : 0 ),
    parameters_( info_.max_chunk_size_*info_.n_parameters_ ),
    prev_parameters_( info_.max_chunk_size_*info_.n_parameters_ ),
    parameters_to_fit_indices_( info_.n_parameters_to_fit_ ),
    user_info_( info_.user_info_size_ ),

    chi_squares_( info_.max_chunk_size_ * info_.n_blocks_per_fit_),
    prev_chi_squares_( info_.max_chunk_size_ ),
    gradients_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_blocks_per_fit_),
    hessians_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_parameters_to_fit_ ),
    deltas_(info_.max_chunk_size_ * info_.n_parameters_to_fit_),

    values_( info_.max_chunk_size_ * info_.n_points_ ),
    derivatives_( info_.max_chunk_size_ * info_.n_points_ * info_.n_parameters_ ),

    lambdas_( info_.max_chunk_size_ ),
    states_( info_.max_chunk_size_ ),
    finished_( info_.max_chunk_size_ ),
    iteration_failed_(info_.max_chunk_size_),
    all_finished_( 1 ),
    n_iterations_( info_.max_chunk_size_ )
{

}

void GPUData::init
(
    int const chunk_size,
    int const chunk_index,
    float const * const data,
    float const * const weights,
    float const * const initial_parameters,
    std::vector<int> const & parameters_to_fit_indices)
{
    chunk_size_ = chunk_size;
    chunk_index_ = chunk_index;

    set(prev_chi_squares_, 0.f, chunk_size_);
    set(states_, 0, chunk_size_);
    set(finished_, 0, chunk_size_);

    write(
        data_,
        &data[chunk_index_*info_.max_chunk_size_*info_.n_points_],
        chunk_size_*info_.n_points_);

    if (info_.use_weights_)
        write(weights_, &weights[chunk_index_*info_.max_chunk_size_*info_.n_points_],
                chunk_size_*info_.n_points_);

    write(
        parameters_,
        &initial_parameters[chunk_index_*info_.max_chunk_size_*info_.n_parameters_],
        chunk_size_ * info_.n_parameters_);

    write(parameters_to_fit_indices_, parameters_to_fit_indices);

    set(lambdas_, 0.001f, chunk_size_);
}

void GPUData::init_user_info(char const * const user_info)
{
    if (info_.user_info_size_ > 0)
        write(user_info_, user_info, info_.user_info_size_);
}

void GPUData::read(bool * dst, int const * src)
{
    int int_dst = 0;
    CUDA_CHECK_STATUS(hipMemcpy(&int_dst, src, sizeof(int), hipMemcpyDeviceToHost));
    * dst = (int_dst == 1) ? true : false;
}

void GPUData::write(float* dst, float const * src, int const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(float), hipMemcpyHostToDevice));
}

void GPUData::write(int* dst, std::vector<int> const & src)
{
    std::size_t const size = src.size() * sizeof(int);
    CUDA_CHECK_STATUS(hipMemcpy(dst, src.data(), size, hipMemcpyHostToDevice));
}

void GPUData::write(char* dst, char const * src, std::size_t const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(char), hipMemcpyHostToDevice));
}

void GPUData::copy(float * dst, float const * src, std::size_t const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(float), hipMemcpyDeviceToDevice));
}

__global__ void set_kernel(int* dst, int const value, int const count)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= count)
        return;

    dst[index] = value;
}

void GPUData::set(int* arr, int const value, int const count)
{
    int const tx = 256;
	int const bx = (count / tx) + 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);

    set_kernel<<< blocks, threads >>>(arr, value, count);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void GPUData::set(int* arr, int const value)
{
    int const tx = 1;
    int const bx = 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);

    set_kernel<<< blocks, threads >>>(arr, value, 1);
    CUDA_CHECK_STATUS(hipGetLastError());
}

__global__ void set_kernel(float* dst, float const value, std::size_t const count)
{
	std::size_t const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= count)
        return;

    dst[index] = value;
}

void GPUData::set(float* arr, float const value, int const count)
{
    int const tx = 256;
	int const bx = (count / tx) + 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);
    set_kernel<<< blocks, threads >>>(arr, value, count);
    CUDA_CHECK_STATUS(hipGetLastError());
}
