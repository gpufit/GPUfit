#include "info.h"
#include <hip/hip_runtime.h>

void Info::get_gpu_properties()
{
    hipDeviceProp_t devProp;
    CUDA_CHECK_STATUS(hipGetDeviceProperties(&devProp, 0));
    max_threads_ = devProp.maxThreadsPerBlock;
    max_blocks_ = devProp.maxGridSize[0];

    std::size_t free_bytes;
    std::size_t total_bytes;
    CUDA_CHECK_STATUS(hipMemGetInfo(&free_bytes, &total_bytes));
    available_gpu_memory_ = std::size_t(double(free_bytes) * 0.1);
    
    if (available_gpu_memory_ > user_info_size_)
    {
        available_gpu_memory_ -= user_info_size_;
    }
    else
    {
        throw std::runtime_error("maximum user info size exceeded");
    }
}

int getDeviceCount()
{
	int deviceCount;
	CUDA_CHECK_STATUS(hipGetDeviceCount(&deviceCount));
	return deviceCount;
}