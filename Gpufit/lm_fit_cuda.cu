#include "hip/hip_runtime.h"
#include "lm_fit.h"
#include <algorithm>
#include "cuda_kernels.cuh"
#include "cuda_gaussjordan.cuh"

void LMFitCUDA::solve_equation_system()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.n_parameters_to_fit_*info_.n_fits_per_block_;
    blocks.x = n_fits_ / info_.n_fits_per_block_;

    cuda_modify_step_widths<<< blocks, threads >>>(
        gpu_data_.hessians_,
        gpu_data_.lambdas_,
        info_.n_parameters_to_fit_,
        gpu_data_.iteration_failed_,
        gpu_data_.finished_,
        info_.n_fits_per_block_);
    CUDA_CHECK_STATUS(hipGetLastError());

    int n_parameters_pow2 = 1;

    while (n_parameters_pow2 < info_.n_parameters_to_fit_)
    {
        n_parameters_pow2 *= 2;
    }

    //set up to run the Gauss Jordan elimination
    int const n_equations = info_.n_parameters_to_fit_;
    int const n_solutions = n_fits_;

    threads.x = n_equations + 1;
    threads.y = n_equations;
    blocks.x = n_solutions;

    //set the size of the shared memory area for each block
    int const shared_size
        = sizeof(float) * ((threads.x * threads.y)
        + n_parameters_pow2 + n_parameters_pow2);

    //set up the singular_test vector
    int * singular_tests;
    CUDA_CHECK_STATUS(hipMalloc((void**)&singular_tests, n_fits_ * sizeof(int)));

    //run the Gauss Jordan elimination
    cuda_gaussjordan<<< blocks, threads, shared_size >>>(
        gpu_data_.deltas_,
        gpu_data_.gradients_,
        gpu_data_.hessians_,
        gpu_data_.finished_,
        singular_tests,
        info_.n_parameters_to_fit_,
        n_parameters_pow2);
    CUDA_CHECK_STATUS(hipGetLastError());

    //set up to update the lm_state_gpu_ variable with the Gauss Jordan results
    threads.x = std::min(n_fits_, 256);
    threads.y = 1;
    blocks.x = int(std::ceil(float(n_fits_) / float(threads.x)));

    //update the lm_state_gpu_ variable
    cuda_update_state_after_gaussjordan<<< blocks, threads >>>(
        n_fits_,
        singular_tests,
        gpu_data_.states_);
    CUDA_CHECK_STATUS(hipGetLastError());

    CUDA_CHECK_STATUS(hipFree(singular_tests));

    threads.x = info_.n_parameters_*info_.n_fits_per_block_;
    threads.y = 1;
    blocks.x = n_fits_ / info_.n_fits_per_block_;

    cuda_update_parameters<<< blocks, threads >>>(
        gpu_data_.parameters_,
        gpu_data_.prev_parameters_,
        gpu_data_.deltas_,
        info_.n_parameters_to_fit_,
        gpu_data_.parameters_to_fit_indices_,
        gpu_data_.finished_,
        info_.n_fits_per_block_);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::calc_curve_values()
{
	dim3  threads(1, 1, 1);
	dim3  blocks(1, 1, 1);

	threads.x = info_.n_points_ * info_.n_fits_per_block_ / info_.n_blocks_per_fit_;
    if (info_.n_blocks_per_fit_ > 1)
        threads.x += info_.n_points_ % threads.x;
	blocks.x = n_fits_ / info_.n_fits_per_block_ * info_.n_blocks_per_fit_;

	cuda_calc_curve_values << < blocks, threads >> >(
		gpu_data_.parameters_,
		n_fits_,
		info_.n_points_,
		info_.n_parameters_,
		gpu_data_.finished_,
		gpu_data_.values_,
		gpu_data_.derivatives_,
		info_.n_fits_per_block_,
        info_.n_blocks_per_fit_,
		info_.model_id_,
		gpu_data_.chunk_index_,
		gpu_data_.user_info_,
		info_.user_info_size_);
	CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::calc_chi_squares()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.power_of_two_n_points_ * info_.n_fits_per_block_ / info_.n_blocks_per_fit_;
    blocks.x = n_fits_ / info_.n_fits_per_block_ * info_.n_blocks_per_fit_;

    int const shared_size = sizeof(float) * threads.x;

    cuda_calculate_chi_squares <<< blocks, threads, shared_size >>>(
        gpu_data_.chi_squares_,
        gpu_data_.states_,
        gpu_data_.data_,
        gpu_data_.values_,
        gpu_data_.weights_,
        info_.n_points_,
        n_fits_,
        info_.estimator_id_,
        gpu_data_.finished_,
        info_.n_fits_per_block_,
        gpu_data_.user_info_,
        info_.user_info_size_);
    CUDA_CHECK_STATUS(hipGetLastError());

    threads.x = std::min(n_fits_, 256);
    blocks.x = int(std::ceil(float(n_fits_) / float(threads.x)));

    if (info_.n_blocks_per_fit_ > 1)
    {
        cuda_sum_chi_square_subtotals <<< blocks, threads >>> (
            gpu_data_.chi_squares_,
            info_.n_blocks_per_fit_,
            n_fits_,
            gpu_data_.finished_);
        CUDA_CHECK_STATUS(hipGetLastError());
    }

    cuda_check_fit_improvement <<< blocks, threads >>>(
        gpu_data_.iteration_failed_,
        gpu_data_.chi_squares_,
        gpu_data_.prev_chi_squares_,
        n_fits_,
        gpu_data_.finished_);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::calc_gradients()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.power_of_two_n_points_ * info_.n_fits_per_block_ / info_.n_blocks_per_fit_;
    blocks.x = n_fits_ / info_.n_fits_per_block_ * info_.n_blocks_per_fit_;

    int const shared_size = sizeof(float) * threads.x;

    cuda_calculate_gradients <<< blocks, threads, shared_size >>>(
        gpu_data_.gradients_,
        gpu_data_.data_,
        gpu_data_.values_,
        gpu_data_.derivatives_,
        gpu_data_.weights_,
        info_.n_points_,
        n_fits_,
        info_.n_parameters_,
        info_.n_parameters_to_fit_,
        gpu_data_.parameters_to_fit_indices_,
        info_.estimator_id_,
        gpu_data_.finished_,
        gpu_data_.iteration_failed_,
        info_.n_fits_per_block_,
        gpu_data_.user_info_,
        info_.user_info_size_);
    CUDA_CHECK_STATUS(hipGetLastError());

    if (info_.n_blocks_per_fit_ > 1)
    {
        int const gradients_size = n_fits_ * info_.n_parameters_to_fit_;
        threads.x = std::min(gradients_size, 256);
        blocks.x = int(std::ceil(float(gradients_size) / float(threads.x)));

        cuda_sum_gradient_subtotals <<< blocks, threads >>> (
            gpu_data_.gradients_,
            info_.n_blocks_per_fit_,
            n_fits_,
            info_.n_parameters_to_fit_,
            gpu_data_.iteration_failed_,
            gpu_data_.finished_);
        CUDA_CHECK_STATUS(hipGetLastError());
    }
}

void LMFitCUDA::calc_hessians()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.n_parameters_to_fit_;
    threads.y = info_.n_parameters_to_fit_;
    blocks.x = n_fits_;

    cuda_calculate_hessians <<< blocks, threads >>>(
        gpu_data_.hessians_,
        gpu_data_.data_,
        gpu_data_.values_,
        gpu_data_.derivatives_,
        gpu_data_.weights_,
        info_.n_points_,
        info_.n_parameters_,
        info_.n_parameters_to_fit_,
        gpu_data_.parameters_to_fit_indices_,
        info_.estimator_id_,
        gpu_data_.iteration_failed_,
        gpu_data_.finished_,
        gpu_data_.user_info_,
        info_.user_info_size_);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::evaluate_iteration(int const iteration)
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = std::min(n_fits_, 256);
    blocks.x = int(std::ceil(float(n_fits_) / float(threads.x)));

    cuda_check_for_convergence<<< blocks, threads >>>(
        gpu_data_.finished_,
        tolerance_,
        gpu_data_.states_,
        gpu_data_.chi_squares_,
        gpu_data_.prev_chi_squares_,
        iteration,
        info_.max_n_iterations_,
        n_fits_);
    CUDA_CHECK_STATUS(hipGetLastError());

    gpu_data_.set(gpu_data_.all_finished_, 1);

    cuda_evaluate_iteration<<< blocks, threads >>>(
        gpu_data_.all_finished_,
        gpu_data_.n_iterations_,
        gpu_data_.finished_,
        iteration,
        gpu_data_.states_,
        n_fits_);
    CUDA_CHECK_STATUS(hipGetLastError());

    gpu_data_.read(&all_finished_, gpu_data_.all_finished_);

    cuda_prepare_next_iteration<<< blocks, threads >>>(
        gpu_data_.lambdas_,
        gpu_data_.chi_squares_,
        gpu_data_.prev_chi_squares_,
        gpu_data_.parameters_,
        gpu_data_.prev_parameters_,
        n_fits_,
        info_.n_parameters_);
    CUDA_CHECK_STATUS(hipGetLastError());
}
